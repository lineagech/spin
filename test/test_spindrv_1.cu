#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <errno.h>
#include <sys/ioctl.h>
#include <sys/types.h>
#include <linux/unistd.h>
//#include <spin/spindrv.h>
#include <spindrv.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/mman.h>
#include <stdarg.h>
#include <unistd.h>
#include <sys/syscall.h>
#include <dlfcn.h>
#include <string>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;
using error = std::runtime_error;

extern void myinit(void);
extern void myend(void);

void transfer_to_address(void *address, size_t size)
{
    CUDA_POINTER_ATTRIBUTE_P2P_TOKENS tokens;
    hipError_t status = hipPointerGetAttribute(&tokens,
            HIP_POINTER_ATTRIBUTE_P2P_TOKENS, (hipDeviceptr_t)address);
    if (hipSuccess == status) {
        // GPU path
        //pass_to_kernel_driver(tokens, address, size);
        fprintf(stderr, "%p is GPU path\n", address);
    }
}

static void getDeviceMemory(void*& bufferPtr, void*& devicePtr, size_t size)
{
    bufferPtr = nullptr;
    devicePtr = nullptr;

    //hipError_t err = hipSetDevice(device);
    hipError_t err = hipSuccess;
    if (err != hipSuccess)
    {
        throw error(string("Failed to set CUDA device: ") + hipGetErrorString(err));
    }

    err = hipMalloc(&bufferPtr, size);
    if (err != hipSuccess)
    {
        throw error(string("Failed to allocate device memory: ") + hipGetErrorString(err));
    }

    err = hipMemset(bufferPtr, 0, size);
    if (err != hipSuccess)
    {
        hipFree(bufferPtr);
        throw error(string("Failed to clear device memory: ") + hipGetErrorString(err));
    }

    hipPointerAttribute_t attrs;
    err = hipPointerGetAttributes(&attrs, bufferPtr);
    if (err != hipSuccess)
    {
        hipFree(bufferPtr);
        throw error(string("Failed to get pointer attributes: ") + hipGetErrorString(err));
    }

    devicePtr = attrs.devicePointer;

    fprintf(stderr, "buff %p, dev %p\n", bufferPtr, devicePtr);
}
int main()
{
    //myinit();
    int fd = open("/dev/spindrv", O_RDONLY);
    spindrv_ioctl_inc_t ioctl_args;
    spindrv_ioctl_param_union send_ioctl;
    memset(&ioctl_args, 0, sizeof(spindrv_ioctl_inc_t));
    
    size_t size = 208*1024*1024;
    void* d_buffer = NULL, *h_buffer = NULL;
    getDeviceMemory(h_buffer, d_buffer, size);
    //hipMalloc(&d_buffer, 1024*1024);
    transfer_to_address(d_buffer, size);



    //hipHostMalloc(&buffer, 1024*1024);
    //h_buffer = malloc(1024*1024);

    //hipMemcpy(d_buffer, h_buffer, 1024*1024, hipMemcpyHostToDevice);
    if (d_buffer == NULL) {
        fprintf(stderr, "cuda d_buffer allocation failed");
        return -1;
    }
    else {
        fprintf(stderr, "d_buffer %p\n", d_buffer);
    }
    ioctl_args.addr = d_buffer;
    ioctl_args.size = size;

    if (fd < 0) {
        fprintf(stderr, "cannot open spindrv\n");
        return -1;
    }
    send_ioctl.set = ioctl_args;

    if (ioctl(fd, SPIN_IOCTL_ADDR_TEST, &send_ioctl) != 0) {
        fprintf(stderr, "Spin Drv IOCTL addr test failed\n");
    }
    hipFree(d_buffer);
    //free(h_buffer);
    //myend();
    close(fd);
    return 0;
}
